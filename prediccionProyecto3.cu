
#include <math.h>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

//Se definen los valores fijos a utilizar en el programa
#define H 288         //Cada bloque manejara 100 datos correspondientes a 5 minutos de mediciones en intervalos de 3 segundos
#define B 2           //Se trabajaran 2 bloques, uno para cada dia
#define VUELTAS 28800 //Cantidad de datos por arreglo
#define N 30          //Varible utilizada en pruebas

using namespace std;

__global__
void inversion(float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < N) y[i] = x[N-1-i];
}

__global__
void raices(float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < N) y[i] = sqrt (x[i]);
}

__global__
void potencia3(float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < N) y[i] = pow ((double)x[i],3.0);
}


__global__
void media(float* arreglo)
{
    float sumatoria = 0;
    float med = 0; //54
    for(int i=0;i<VUELTAS;i++){
  	  sumatoria = sumatoria + arreglo[i];	  
    }
    med = sumatoria/(float) VUELTAS;
    sumatoria = med; 
}

//Subrutina que calcula cual fue la mayor medicion en el dia con hora a la que fue medida
__global__
void mayor(float* arreglo){
    float may=arreglo[0];
    for(int i=0;i<VUELTAS;i++)
    { if(arreglo[i]>may){
      may=arreglo[i];}             
    }
}

//Subrutina que calcula cual fue la menor medicion en el dia con hora a la que fue medida
__global__
void menor(float* arreglo){
    float men=arreglo[0];
    for(int i=0;i<VUELTAS;i++)
    { if(arreglo[i]<men){
      men=arreglo[i];}
       
    } 
}

//Subrutina que calcula la prediccion de datos para un dia siguiente a traves de la regresion lineal de un tipo de medicion hecha por cada 5 minutos en intervalos de 3 segundos
__global__
void prediccion(float* arreglo, float* salida){
        int i = blockIdx.x*blockDim.x + threadIdx.x;
        int q = 0;
	float k = 100.0;
	float m = 0;
	float sumatoria = 0;
	float sumasDif = 0;
	float potencia = 0;
	float pendiente = 0;
	//float nueva = 0;
        q = i*100;
	for(int j = q; j<q+100; j++){
		sumatoria = sumatoria + arreglo[j];
	}
	sumatoria = sumatoria/k;
	for(int j = q; j<q+100; j++){
		sumasDif = arreglo[j] - sumatoria;
	}
	potencia = (float)pow((double)sumasDif,2.00);
	pendiente = potencia/k;

	for(int j = q; j<q+100; j++){
		salida[j] = sumatoria + pendiente*m;
		m = m + 1;
	}

}

//Inicio del programa
int main(void)
{
  // declaraciones de componentes CUDA, Streams y memoria
  hipStream_t stream1, stream2, stream3, stream4, stream5, stream6;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  hipStreamCreate(&stream3);
  hipStreamCreate(&stream4);
  hipStreamCreate(&stream5);
  hipStreamCreate(&stream6);

  //Se abren los archivos y se limpian
  ofstream ArchivoPrediccion("181113_estCU.csv");
  ArchivoPrediccion.close();
  ofstream ArchivoPrediccion2("181114_estCU.csv");
  ArchivoPrediccion2.close();

  //Se crean los vectores que guardaran los string de horas de los archivos .csv
  string horas[VUELTAS];
  string horas2[VUELTAS];

  //Se inician las variables que guardaran los tiempos de ejecucion de cada kernel
  float milliseconds1 = 0;
  float milliseconds2 = 0;
  float milliseconds3 = 0;
  float milliseconds4 = 0;
  float milliseconds5 = 0;
  float milliseconds6 = 0;

  //Se crean las variables de vectores que llevaran datos y compiaran entre el host y el device
  float *vectorTemperatura1, *vectorHumedad1, *vectorPresion1, *res_stream1, *res_stream2, *res_stream3;
  float *vectorTemperatura2, *vectorHumedad2, *vectorPresion2, *res_stream4, *res_stream5, *res_stream6;
  float *dev_res1, *dev_res2, *dev_res3;
  float *dev_res4, *dev_res5, *dev_res6;
  // reserva en el host

  // reserva en el device
  hipMalloc( (void**)&dev_res1, VUELTAS*sizeof(float));
  hipMalloc( (void**)&dev_res2, VUELTAS*sizeof(float));
  hipMalloc( (void**)&dev_res3, VUELTAS*sizeof(float));
  hipMalloc( (void**)&dev_res4, VUELTAS*sizeof(float));
  hipMalloc( (void**)&dev_res5, VUELTAS*sizeof(float));
  hipMalloc( (void**)&dev_res6, VUELTAS*sizeof(float));

  //Asignacion de memoria al host
  hipHostAlloc((void**)&vectorTemperatura1,VUELTAS*sizeof(float),hipHostMallocDefault);
  hipHostAlloc((void**)&vectorHumedad1,VUELTAS*sizeof(float),hipHostMallocDefault);
  hipHostAlloc((void**)&vectorPresion1,VUELTAS*sizeof(float),hipHostMallocDefault);
  hipHostAlloc((void**)&vectorTemperatura2,VUELTAS*sizeof(float),hipHostMallocDefault);
  hipHostAlloc((void**)&vectorHumedad2,VUELTAS*sizeof(float),hipHostMallocDefault);
  hipHostAlloc((void**)&vectorPresion2,VUELTAS*sizeof(float),hipHostMallocDefault);

  hipHostAlloc((void**)&res_stream1,VUELTAS*sizeof(float),hipHostMallocDefault);
  hipHostAlloc((void**)&res_stream2,VUELTAS*sizeof(float),hipHostMallocDefault);
  hipHostAlloc((void**)&res_stream3,VUELTAS*sizeof(float),hipHostMallocDefault);
  hipHostAlloc((void**)&res_stream4,VUELTAS*sizeof(float),hipHostMallocDefault);
  hipHostAlloc((void**)&res_stream5,VUELTAS*sizeof(float),hipHostMallocDefault);
  hipHostAlloc((void**)&res_stream6,VUELTAS*sizeof(float),hipHostMallocDefault);

  // se crean los eventos
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

/////////////////////////////////////////////////////////////////////////////////////////////////

  // Inicializacion de datos por lectura de archivos .csv
  // Se leen los datos del dia 1
  ifstream datos("181113.csv");
  string linea;
  int contadorPosicion = 0;

  // Se obtienen los datos separados de cada linea guardada
  while(getline(datos,linea)){
       string delimiter = ";";
       size_t pos = 0;
       string token;
       int cont = 0;
       while ((pos = linea.find(delimiter)) != std::string::npos) {
          token = linea.substr(0, pos);
          linea.erase(0, pos + delimiter.length());
          if(cont == 0){
	     horas[contadorPosicion] = token;
          }
	  if(cont == 1){
	     vectorTemperatura1[contadorPosicion] = (float)(::atof(token.c_str()));
          }
	  if(cont == 2){
	     vectorHumedad1[contadorPosicion] = (float)(::atof(token.c_str()));
          }
	  if(cont == 3){
	     vectorPresion1[contadorPosicion] =  (float)(::atof(token.c_str()));
          }
	  cont = cont + 1;

       }
       contadorPosicion = contadorPosicion + 1;
  }


////////////////////////////////////////////////////////////////////////////////

  //Se ejecutan 3 kernels cada uno en un stream diferente y haciendolo en 288 bloques cada uno, de manera aplicar regresion lineal cada 100 datos equivalente a 5 minutos de mediciones para el dia 1
  for(int i=0;i < H;i++){
  // copia de datos hacia el device
  hipMemcpyAsync(dev_res1, vectorTemperatura1, VUELTAS*sizeof(float), hipMemcpyHostToDevice,stream1);

  //Se hace la medicion del tiempo atraves de events
  hipEventRecord(start);
  prediccion<<<1, H>>>(vectorTemperatura1, dev_res1);
  hipEventRecord(stop);

  hipMemcpyAsync(res_stream1, dev_res1, VUELTAS*sizeof(float), hipMemcpyDeviceToHost,stream1);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds1, start, stop);

/////////////////////////////////////////////////////////////////////////////

  hipMemcpyAsync(dev_res2, vectorHumedad1, VUELTAS*sizeof(float), hipMemcpyHostToDevice,stream2);

  //Se hace la medicion del tiempo atraves de events
  hipEventRecord(start);
  prediccion<<<1, H>>>(vectorHumedad1, dev_res2);
  hipEventRecord(stop);

  hipMemcpyAsync(res_stream2, dev_res2, VUELTAS*sizeof(float), hipMemcpyDeviceToHost, stream2);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds2, start, stop);

////////////////////////////////////////////////////////////////////////////////

  hipMemcpyAsync(dev_res3, vectorPresion1, VUELTAS*sizeof(float), hipMemcpyHostToDevice,stream3);

  //Se hace la medicion del tiempo atraves de events
  hipEventRecord(start);
  prediccion<<<1, H>>>(vectorPresion1, dev_res3);
  hipEventRecord(stop);

  hipMemcpyAsync(res_stream3, dev_res3, VUELTAS*sizeof(float), hipMemcpyDeviceToHost,stream3);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds3, start, stop);

  }

///////////////////////////////////////////////////////////////////////////////

  //Se sincronizan los streams
  hipStreamSynchronize(stream1); // wait for stream1 to finish
  hipStreamSynchronize(stream2); // wait for stream2 to finish
  hipStreamSynchronize(stream3); // wait for stream3 to finish

/////////////////////////////////////////////////////////////////////////////////////////////////////////////
  // Se leen los datos del dia 2
  ifstream datos2("181114.csv");
  contadorPosicion = 0;

  // Se obtienen los datos separados de cada linea guardada
  while(getline(datos2,linea)){
       string delimiter = ";";
       size_t pos = 0;
       string token;
       int cont = 0;
       while ((pos = linea.find(delimiter)) != std::string::npos) {
          token = linea.substr(0, pos);
          linea.erase(0, pos + delimiter.length());
          if(cont == 0){
	     horas2[contadorPosicion] = token;
          }
	  if(cont == 1){
	     vectorTemperatura2[contadorPosicion] = (float)(::atof(token.c_str()));
          }
	  if(cont == 2){
	     vectorHumedad2[contadorPosicion] = (float)(::atof(token.c_str()));
          }
	  if(cont == 3){
	     vectorPresion2[contadorPosicion] =  (float)(::atof(token.c_str()));
          }
	  cont = cont + 1;

       }
       contadorPosicion = contadorPosicion + 1;
  }


////////////////////////////////////////////////////////////////////////////////

  //Se ejecutan 3 kernels cada uno en un stream diferente y haciendolo en 288 bloques cada uno, de manera aplicar regresion lineal cada 100 datos equivalente a 5 minutos de mediciones para el dia 2
  for(int i=0;i < H;i++){
  // copia de datos hacia el device
  hipMemcpyAsync(dev_res4, vectorTemperatura2, VUELTAS*sizeof(float), hipMemcpyHostToDevice,stream4);

  //Se hace la medicion del tiempo atraves de events
  hipEventRecord(start);
  prediccion<<<1, H>>>(vectorTemperatura2, dev_res4);
  hipEventRecord(stop);

  hipMemcpyAsync(res_stream4, dev_res4, VUELTAS*sizeof(float), hipMemcpyDeviceToHost,stream4);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds4, start, stop);

/////////////////////////////////////////////////////////////////////////////

  hipMemcpyAsync(dev_res5, vectorHumedad2, VUELTAS*sizeof(float), hipMemcpyHostToDevice,stream5);

  //Se hace la medicion del tiempo atraves de events
  hipEventRecord(start);
  prediccion<<<1, H>>>(vectorHumedad2, dev_res5);
  hipEventRecord(stop);

  hipMemcpyAsync(res_stream5, dev_res5, VUELTAS*sizeof(float), hipMemcpyDeviceToHost, stream5);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds5, start, stop);


////////////////////////////////////////////////////////////////////////////////

  hipMemcpyAsync(dev_res6, vectorPresion2, VUELTAS*sizeof(float), hipMemcpyHostToDevice,stream6);

  //Se hace la medicion del tiempo atraves de events
  hipEventRecord(start);
  prediccion<<<1, H>>>(vectorPresion2, dev_res6);
  hipEventRecord(stop);

  hipMemcpyAsync(res_stream6, dev_res6, VUELTAS*sizeof(float), hipMemcpyDeviceToHost,stream6);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds6, start, stop);

  }

///////////////////////////////////////////////////////////////////////////////

  //Se sincronizan los streams
  hipStreamSynchronize(stream4); // wait for stream1 to finish
  hipStreamSynchronize(stream5); // wait for stream2 to finish
  hipStreamSynchronize(stream6); // wait for stream3 to finish

/////////////////////////////////////////////////////////////////////////////////
//Se guardan los datos predecidos en un archivo csv correspondiente
  ofstream Archivo("181113_estCU.csv");
  for(int i=0;i<VUELTAS;i++){
	Archivo << horas[i] << ";" << res_stream1[i] << ";" << res_stream2[i] << ";" << res_stream3[i] << ";" << endl;      
  }
  Archivo.close();

  ofstream Archivo2("181114_estCU.csv");
  for(int i=0;i<VUELTAS;i++){
	Archivo2 << horas2[i] << ";" << res_stream4[i] << ";" << res_stream5[i] << ";" << res_stream6[i] << ";" << endl;      
  }
  Archivo2.close();

  //Se imprimen los tiempos que tardaron cada uno de los kernels
  printf("Tiempo del kernel para la prediccion de temperaturas del dia 1: %f milisegundos\n", milliseconds1);
  printf("Tiempo del kernel para la prediccion de humedades del dia 1: %f milisegundos\n", milliseconds2);
  printf("Tiempo del kernel para la prediccion de presiones del dia 1: %f milisegundos\n", milliseconds3);
  printf("Tiempo del kernel para la prediccion de temperaturas del dia 2: %f milisegundos\n", milliseconds4);
  printf("Tiempo del kernel para la prediccion de humedades del dia 2: %f milisegundos\n", milliseconds5);
  printf("Tiempo del kernel para la prediccion de presiones del dia 2: %f milisegundos\n", milliseconds6);

  //Se destruyen todos los componentes CUDA y se libera la memoria
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipStreamDestroy(stream3);
  hipStreamDestroy(stream4);
  hipStreamDestroy(stream5);
  hipStreamDestroy(stream6);

  hipFree(dev_res1);
  hipFree(dev_res2);
  hipFree(dev_res3);
  hipFree(dev_res4);
  hipFree(dev_res5);
  hipFree(dev_res6);

  // salida
  printf("\npulsa INTRO para finalizar...");
  fflush(stdin);
  char tecla = getchar();
  return 0;
}
